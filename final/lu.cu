#include "hip/hip_runtime.h"
#include<iostream>
#include<stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <cusp/detail/lu.h>
#include <cusp/lapack/lapack.h>

#define N 16

void initializeArray(double* arr, int nElements)
{
    const int myMin = -5;
    const int myMax = 5;
    srand(11235);

    for( int i=0; i<nElements; i++)
	{
		for( int j=0; j<nElements; j++)
		{
        		arr[(i*nElements) + j] = (double)(rand()/((double)RAND_MAX) * (myMax-myMin) + myMin);
		}		
	}
}

__global__ void LUdecomp(double *A, double *LU) {
   
	 int i = blockIdx.x*blockDim.x + threadIdx.x;
	LU[i] = A[i];

}

int main() {
  int size = N * sizeof(double); 

  double *matA;
  hipHostMalloc(&matA, size*N);

  double *matLU;
  hipHostMalloc(&matLU, size*N);

  initializeArray(matA, N);

	//Start inclusive timing here
	hipEvent_t startIn, stopIn;
	hipEventCreate(&startIn);
	hipEventCreate(&stopIn);
	hipEventRecord(startIn, 0);

	double *d_matA;  hipMalloc(&d_matA, size*N);
	double *d_matLU; hipMalloc(&d_matLU, size*N);

	hipMemcpy(d_matA, matA, N*size, hipMemcpyHostToDevice);
	hipMemcpy(d_matLU, matLU, N*size, hipMemcpyHostToDevice);

	//Start exclusive timing here
	hipEvent_t startEx, stopEx;
	hipEventCreate(&startEx);
	hipEventCreate(&stopEx);
	hipEventRecord(startEx, 0);

  LUdecomp<<<1,256>>>(d_matA, d_matLU);

	//Stop exclusive timing here
	hipEventRecord(stopEx, 0);
	hipEventSynchronize(stopEx);
	float exTime;
	hipEventElapsedTime(&exTime, startEx, stopEx);
	hipEventDestroy(startEx);
	hipEventDestroy(stopEx);

	hipMemcpy(matLU, d_matLU, size*N, hipMemcpyDeviceToHost);

	//Stop inclusive timing here
	hipEventRecord(stopIn, 0);
	hipEventSynchronize(stopIn);
	float inTime;
	hipEventElapsedTime(&inTime, startIn, stopIn);
	hipEventDestroy(startIn);
	hipEventDestroy(stopIn);

	//Output timing
	printf("Inclusive time: %f ms. \n", inTime);
	printf("Exclusive time: %f ms. \n", exTime);

	// For verification, just to output small matrices to test
	FILE *fpA, *fpL;
	fpA = fopen("./bin/matAd.inp","w");
	fpL = fopen("./bin/matLUd.inp","w");
    for( int i=0; i<N; i++)
	{
		for( int j=0; j<N; j++)
		{
			fprintf(fpA, "%f ", matA[(i*N) + j]);
			fprintf(fpL, "%f ", matLU[(i*N) + j]);
		}		
		fprintf(fpA, "\n");
		fprintf(fpL, "\n");
	}

  //free resources
  hipFree(matLU); hipFree(matA);
  hipFree(d_matLU);  hipFree(d_matA);
  return 0;
}	
