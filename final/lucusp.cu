#include<iostream>
#include<stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <cusp/array2d.h>
#include <cusp/print.h>
#include <cusp/krylov/cg.h>
#include <cusp/io/matrix_market.h>
#include <cusp/precond/diagonal.h>

#define N 16


int main() {

	cusp::array2d<double, cusp::host_memory> matA(N,N);
	cusp::array1d<double, cusp::host_memory> vecb(N);

	//Initialize the array
    const int myMin = -5;
    const int myMax = 5;
    srand(11235);
    for( int i=0; i<N; i++)
	{
		for( int j=0; j<N; j++)
		{
        		matA(i,j) = (double)(rand()/((double)RAND_MAX) * (myMax-myMin) + myMin);
		}		
	}
	for( int j=0; j<N; j++)
	{
   		vecb[j] = (double)(rand()/((double)RAND_MAX) * (myMax-myMin) + myMin);
	}	

	//Write matA to MM format
	cusp::io::write_matrix_market_file(matA, "./bin/A.mtx");
    // load A from disk into a csr_matrix
    cusp::csr_matrix<int, double, cusp::device_memory> d_matB;
    cusp::io::read_matrix_market_file(d_matB, "./bin/A.mtx");

	//Start inclusive timing here
	hipEvent_t startIn, stopIn;
	hipEventCreate(&startIn);
	hipEventCreate(&stopIn);
	hipEventRecord(startIn, 0);

	//cusp::array2d<double, cusp::device_memory> d_matA(matA);
    cusp::array1d<double, cusp::device_memory>   d_vecb(vecb);
	cusp::array1d<double, cusp::device_memory> x(N, 0);

        // set stopping criteria (iteration_limit = 100, relative_tolerance = 1e-6)
        cusp::verbose_monitor<double> monitor(d_vecb, 10000, 1e-6);
        // setup preconditioner
        //cusp::precond::diagonal<double, cusp::device_memory> M(d_matB);
    cusp::krylov::cg(d_matB, x, d_vecb, monitor);

	//Start exclusive timing here
	hipEvent_t startEx, stopEx;
	hipEventCreate(&startEx);
	hipEventCreate(&stopEx);
	hipEventRecord(startEx, 0);


	//Stop exclusive timing here
	hipEventRecord(stopEx, 0);
	hipEventSynchronize(stopEx);
	float exTime;
	hipEventElapsedTime(&exTime, startEx, stopEx);
	hipEventDestroy(startEx);
	hipEventDestroy(stopEx);

	//Stop inclusive timing here
	hipEventRecord(stopIn, 0);
	hipEventSynchronize(stopIn);
	float inTime;
	hipEventElapsedTime(&inTime, startIn, stopIn);
	hipEventDestroy(startIn);
	hipEventDestroy(stopIn);

	// For verification, just to output small matrices to test
	//cusp::print(matA);

	//Output timing
	printf("Inclusive time: %f ms. \n", inTime);
	printf("Exclusive time: %f ms. \n", exTime);

  return 0;
}	
