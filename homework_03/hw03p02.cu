/*
 * hw03p02.cu
 *
 *  Created on: Oct 02, 2015
 *      Author: Kazi
 *  Usage:
 * 	
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>


/* 
 */
__global__ void sumArray(double* a, double* b, double* c, int size)
{
	int entry = threadIdx.x + blockIdx.x*blockDim.x;
	
	//Perform the addition
	if (entry < size)
	{
		c[entry] = a[entry] + b[entry];
	}
}

/*
 */
int main(int argc, char* argv[])
{
	//Set the size of the arrays, threads, blocks
	int length = (1 << 20); //Input exponent of 2 on the right
	int threads = 1024;
	int blocks = length/threads;
	printf("The length is: %d\n", length);

	//Allocate memory on the host
	double *hA = (double*)malloc(length*sizeof(double));
	double *hB = (double*)malloc(length*sizeof(double));
	double *hC = (double*)malloc(length*sizeof(double));
	double *refC = (double*)malloc(length*sizeof(double));

	//Allocate memory on the GPU
	double *dA, *dB, *dC;
	hipMalloc((void**) &dA, sizeof(double)*length);
	hipMalloc((void**) &dB, sizeof(double)*length);
	hipMalloc((void**) &dC, sizeof(double)*length);

	//Generate the random arrays
	int i;
	double temp;
	for(i=0; i<length; i++)
	{
		temp = ((double)rand())/(RAND_MAX/20)-10;
		hA[i] = temp;
		temp = ((double)rand())/(RAND_MAX/20)-10;
		hB[i] = temp;
	}

	//Add those arrays together on the cpu, for a reference
	int j;
	for(j=0; j<length; j++)
	{
		//*(refC+j) = *(hA+j) + *(hB+j);
		refC[j] = hA[j] + hB[j];
		//printf("%lf\n", refC[j]);
	}

	//Start inclusive timing here
	hipEvent_t startIn, stopIn;
	hipEventCreate(&startIn);
	hipEventCreate(&stopIn);
	hipEventRecord(startIn, 0);

	//Copy hA,hB onto dA,dB
	hipMemcpy(dA, hA, sizeof(double)*length, hipMemcpyHostToDevice);
	hipMemcpy(dB, hB, sizeof(double)*length, hipMemcpyHostToDevice);

	//Start exclusive timing here
	hipEvent_t startEx, stopEx;
	hipEventCreate(&startEx);
	hipEventCreate(&stopEx);
	hipEventRecord(startEx, 0);

	//Use kernel to sum the two arrays
	sumArray <<<blocks,threads>>> (dA, dB, dC, length);

	//Stop exclusive timing here
	hipEventRecord(stopEx, 0);
	hipEventSynchronize(stopEx);
	float exTime;
	hipEventElapsedTime(&exTime, startEx, stopEx);
	hipEventDestroy(startEx);
	hipEventDestroy(stopEx);

	//Copy dC back into hC
	hipMemcpy(hC, dC, sizeof(double)*length, hipMemcpyDeviceToHost);

	//Stop inclusive timing here
	hipEventRecord(stopIn, 0);
	hipEventSynchronize(stopIn);
	float inTime;
	hipEventElapsedTime(&inTime, startIn, stopIn);
	hipEventDestroy(startIn);
	hipEventDestroy(stopIn);

	//For testing
	int k;
	for(k=0; k<length; k++)
	{
		//printf("%lf\n", hC[k]);
	}

	printf("Inclusive time: %f ms. \n", inTime);
	printf("Exclusive time: %f ms. \n", exTime);

	//Cleanup
	if(dA) hipFree(dA);
	if(dB) hipFree(dB);
	if(hA) free(hA);
	if(hB) free(hB);
	if(hC) free(hC);
	if(refC) free(refC);

	return 0;
}
