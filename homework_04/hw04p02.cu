#include "hip/hip_runtime.h"
/*
 * hw04p02.cu
 *
 *  Created on: Oct 04, 2015
 *      Author: Kazi
 *  Usage:
 * 	It performs integer multiplication of a 16x32 matrix with a 32x1 vector
 *	on a GPU. Does not take any arguments. Just generates predefined matrices
 *	and reports the time taken to do the multiplication.
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <iostream>


/* 
 * Given an array A of size (h x w) and a vector b of size (w), it takes the product
 * Ab and writes it to a vector c of size (h).
 */
__global__ void multArray(int* A, int* b, int* c, int w, int size)
{
	int ti = threadIdx.x;

	int dsum = 0; //The sum for the dot product
	int k;
	//Perform the multiplication
	if (ti < size)
	{
		for(k=0; k<w; k++)
		{
			//Take the dot of a row of A with b
			dsum = dsum + A[ti*w+k]*b[k];
		}
		c[ti] = dsum;
	}
}

/*
 * Entry point for the program. Currently specifies matrix and vector size.
 * Allocates memory on the host and device and then creates matrices on the
 * host. Copies them over to the device to multiply them. Copies the result
 * back over to the host.
 */
int main(int argc, char* argv[])
{
	//Set the size of the arrays, threads, blocks
	int height = 16;
	int width = 32;
	int threads = height;
	int blocks = 1;

	//Allocate memory on the host
	int *hA = (int*)malloc(width*height*sizeof(int));
	int *hb = (int*)malloc(width*sizeof(int));
	int *hc = (int*)malloc(height*sizeof(int));

	//Allocate memory on the device
	int *dA, *db, *dc;
	hipMalloc((void**) &dA, sizeof(int)*width*height);
	hipMalloc((void**) &db, sizeof(int)*width);
	hipMalloc((void**) &dc, sizeof(int)*height);

	//Generate the matrices on the host
	int i;
	int j;
	for(i=0; i<height; i++)
	{
		hc[i] = 0; //Let the storage array be all zeros
		for(j=0; j<width; j++)
		{
			hA[i*width + j] = i+j; //Set the matrix A
			if (i == 0) hb[j] = j; //Set the vector b
		}
	}

	//Start inclusive timing here
	hipEvent_t startIn, stopIn;
	hipEventCreate(&startIn);
	hipEventCreate(&stopIn);
	hipEventRecord(startIn, 0);

	//Copy hA,hb, hc onto dA,db, dc
	hipMemcpy(dA, hA, sizeof(int)*width*height, hipMemcpyHostToDevice);
	hipMemcpy(db, hb, sizeof(int)*width, hipMemcpyHostToDevice);
	hipMemcpy(dc, hc, sizeof(int)*width, hipMemcpyHostToDevice);

	//Start exclusive timing here
	hipEvent_t startEx, stopEx;
	hipEventCreate(&startEx);
	hipEventCreate(&stopEx);
	hipEventRecord(startEx, 0);

	//Use kernel to multiply A and b
	multArray <<<blocks,threads>>> (dA, db, dc, width, width*height);

	//Stop exclusive timing here
	hipEventRecord(stopEx, 0);
	hipEventSynchronize(stopEx);
	float exTime;
	hipEventElapsedTime(&exTime, startEx, stopEx);
	hipEventDestroy(startEx);
	hipEventDestroy(stopEx);

	//Copy dc back into hc
	hipMemcpy(hc, dc, sizeof(int)*height, hipMemcpyDeviceToHost);

	//Stop inclusive timing here
	hipEventRecord(stopIn, 0);
	hipEventSynchronize(stopIn);
	float inTime;
	hipEventElapsedTime(&inTime, startIn, stopIn);
	hipEventDestroy(startIn);
	hipEventDestroy(stopIn);

	//For testing - to see what the result vector looks like
	for(j=0; j<height; j++)
	{
		//printf("%d\n", hc[j]);
		std::cout << j << ": " << hc[j] << std::endl;
	}

	//Output timing
	printf("Inclusive time: %f ms. \n", inTime);
	printf("Exclusive time: %f ms. \n", exTime);

	//Get device properties
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	printf("Device name: %s \n", deviceProp.name);
	printf("Clock rate: %d \n", deviceProp.clockRate);
	printf("Multiprocessors: %d \n", deviceProp.multiProcessorCount);
	printf("L2 Cache: %d \n", deviceProp.l2CacheSize);
	printf("Max threads per MP: %d \n", deviceProp.maxThreadsPerMultiProcessor);
	printf("Warp size: %d \n", deviceProp.warpSize);
	printf("Max threads per block: %d \n", deviceProp.maxThreadsPerBlock);
	printf("Max registers per block: %d \n", deviceProp.regsPerBlock);
	printf("Max blocks per MP: 32 \n"); //From table
	printf("Max warps per MP: 64 \n"); //From table
	printf("Shared memory per block (B): %d \n", deviceProp.sharedMemPerBlock);
	printf("Compute capability: %d.%d.\n", deviceProp.major, deviceProp.minor);

	//Write to file
	FILE *fp;
	fp = fopen("./problem2.out","w");
	fprintf(fp, "Results of hw04p02.cu: \n");
	for (i=0; i<height; i++)
	{
		fprintf(fp, "%d\n", hc[i]);
	}
	fclose(fp);

	//Cleanup
	if(dA) hipFree(dA);
	if(db) hipFree(db);
	if(dc) hipFree(dc);
	if(hA) free(hA);
	if(hb) free(hb);
	if(hc) free(hc);

	return 0;
}
