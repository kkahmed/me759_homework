#include<iostream>
#include<stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_runtime_api.h>

#define N 1000000
#define RADIUS 3

int checkResults(int startElem, int endElem, float* cudaRes, float* res)
{
    int nDiffs=0;
    const float smallVal = 0.000001f;
    for(int i=startElem; i<endElem; i++)
        if(fabs(cudaRes[i]-res[i])>smallVal)
            nDiffs++;
    return nDiffs;
}

void initializeWeights(float* weights, int rad)
{
    // for now hardcoded for RADIUS=3
    weights[0] = 0.50f;
    weights[1] = 0.75f;
    weights[2] = 1.25f;
    weights[3] = 2.00f;
    weights[4] = 1.25f;
    weights[5] = 0.75f;
    weights[6] = 0.50f;
}

void initializeArray(float* arr, int nElements)
{
    const int myMinNumber = -5;
    const int myMaxNumber = 5;
    srand(time(NULL));
    for( int i=0; i<nElements; i++)
        arr[i] = (float)(rand() % (myMaxNumber - myMinNumber + 1) + myMinNumber);
}

void applyStencil1D_SEQ(int sIdx, int eIdx, const float *weights, float *in, float *out) {
  
  for (int i = sIdx; i < eIdx; i++) {   
    out[i] = 0;
    //loop over all elements in the stencil
    for (int j = -RADIUS; j <= RADIUS; j++) {
      out[i] += weights[j + RADIUS] * in[i + j]; 
    }
    out[i] = out[i] / (2 * RADIUS + 1);
  }
}

__global__ void applyStencil1D(int sIdx, int eIdx, const float *weights, float *in, float *out) {
    int i = sIdx + blockIdx.x*blockDim.x + threadIdx.x;

    if( i < eIdx ) {
        float result = 0.f;
        result += weights[0]*in[i-3];
        result += weights[1]*in[i-2];
        result += weights[2]*in[i-1];
        result += weights[3]*in[i];
        result += weights[4]*in[i+1];
        result += weights[5]*in[i+2];
        result += weights[6]*in[i+3];
        result /=7.f;
        out[i] = result;
    }
}

int main() {
  int size = N * sizeof(float); 
  int wsize = (2 * RADIUS + 1) * sizeof(float); 
  //allocate resources
  float *weights = (float *)malloc(wsize);
  float *in      = (float *)malloc(size);
  float *out     = (float *)malloc(size); 
  float *cuda_out= (float *)malloc(size); 
  initializeWeights(weights, RADIUS);
  initializeArray(in, N);

	//Start inclusive timing here
	hipEvent_t startIn, stopIn;
	hipEventCreate(&startIn);
	hipEventCreate(&stopIn);
	hipEventRecord(startIn, 0);
  float *d_weights;  hipMalloc(&d_weights, wsize);
  float *d_in;       hipMalloc(&d_in, size);
  float *d_out;      hipMalloc(&d_out, size);
  
  hipMemcpy(d_weights,weights,wsize,hipMemcpyHostToDevice);
  hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);
	//Start exclusive timing here
	hipEvent_t startEx, stopEx;
	hipEventCreate(&startEx);
	hipEventCreate(&stopEx);
	hipEventRecord(startEx, 0);
  applyStencil1D<<<(N+511)/512, 512>>>(RADIUS, N-RADIUS, d_weights, d_in, d_out);
	//Stop exclusive timing here
	hipEventRecord(stopEx, 0);
	hipEventSynchronize(stopEx);
	float exTime;
	hipEventElapsedTime(&exTime, startEx, stopEx);
	hipEventDestroy(startEx);
	hipEventDestroy(stopEx);
  hipMemcpy(cuda_out, d_out, size, hipMemcpyDeviceToHost);
	//Stop inclusive timing here
	hipEventRecord(stopIn, 0);
	hipEventSynchronize(stopIn);
	float inTime;
	hipEventElapsedTime(&inTime, startIn, stopIn);
	hipEventDestroy(startIn);
	hipEventDestroy(stopIn);

	//Start cpu timing here
	hipEvent_t startCPU, stopCPU;
	hipEventCreate(&startCPU);
	hipEventCreate(&stopCPU);
	hipEventRecord(startCPU, 0);
  applyStencil1D_SEQ(RADIUS, N-RADIUS, weights, in, out);
	//Stop cpu timing here
	hipEventRecord(stopCPU, 0);
	hipEventSynchronize(stopCPU);
	float cpuTime;
	hipEventElapsedTime(&cpuTime, startCPU, stopCPU);
	hipEventDestroy(startCPU);
	hipEventDestroy(stopCPU);

	//Output timing
	printf("Inclusive time: %f ms. \n", inTime);
	printf("Exclusive time: %f ms. \n", exTime);
	printf("CPU time: %f ms. \n", cpuTime);

  int nDiffs = checkResults(RADIUS, N-RADIUS, cuda_out, out);
  nDiffs==0? std::cout<<"Looks good.\n": std::cout<<"Doesn't look good: " << nDiffs << "differences\n";

  //free resources
  free(weights); free(in); free(out); free(cuda_out);
  hipFree(d_weights);  hipFree(d_in);  hipFree(d_out);
  return 0;
}
